#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#define TOTAL_SIZE 1024
//#define TOTAL_SIZE (1024*1024*1024)
#define block_dim 1024
#define chk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

double *h_c, *h_a, *h_b;

double **d_c, **d_a, **d_b;

hipStream_t *streams;

hipEvent_t start, finish;


void allocate(int devices, int multi_gpu) {

    int i = 0, parts, rem;

    //h_c = (double *) malloc(sizeof(double) * TOTAL_SIZE);
    //h_a = (double *) malloc(sizeof(double) * TOTAL_SIZE);
    //h_b = (double *) malloc(sizeof(double) * TOTAL_SIZE);

    d_c = (double **) malloc(sizeof(double *) * devices);
    d_a = (double **) malloc(sizeof(double *) * devices);
    d_b = (double **) malloc(sizeof(double *) * devices);

    hipHostMalloc((void **) &h_c, sizeof(double) * TOTAL_SIZE, hipHostMallocDefault);
    hipHostMalloc((void **) &h_a, sizeof(double) * TOTAL_SIZE, hipHostMallocDefault);
    hipHostMalloc((void **) &h_b, sizeof(double) * TOTAL_SIZE, hipHostMallocDefault);
    
    //cudaMallocHost((void **)d_c, sizeof(double *) * devices);
    //cudaMallocHost((void **)d_a, sizeof(double *) * devices);
    //cudaMallocHost((void **)d_b, sizeof(double *) * devices);

    streams = (hipStream_t *) malloc(sizeof(hipStream_t) * devices);
    
    for (i=0; i<devices; ++i) {
        hipStreamCreate(&streams[i]);
    }

    parts = TOTAL_SIZE / devices;
    rem = TOTAL_SIZE % devices;
    i = 0;

    if (multi_gpu) {
        for (i=0; i<devices-1; ++i) {
    
            hipSetDevice(i);
            printf("\nS%d", streams[i]);
            chk(hipMallocAsync((void **) &d_c[i], sizeof(double) * parts, streams[i]));
            chk(hipMallocAsync((void **) &d_a[i], sizeof(double) * parts, streams[i]));
            chk(hipMallocAsync((void **) &d_b[i], sizeof(double) * parts, streams[i]));
        }
    }

    hipSetDevice(i);
    chk(hipMallocAsync((void **) &d_c[i], sizeof(double) * (parts + rem), streams[i]));
    chk(hipMallocAsync((void **) &d_a[i], sizeof(double) * (parts + rem), streams[i]));
    chk(hipMallocAsync((void **) &d_b[i], sizeof(double) * (parts + rem), streams[i]));

    hipEventCreate(&start);
    hipEventCreate(&finish);            
}

extern "C" __global__ void vec_add(double *c, double *a, double *b, int PART_SIZE) {
    
    int t = threadIdx.x + blockIdx.x * blockDim.x;

    if (t < TOTAL_SIZE && t < PART_SIZE) {
    
        c[t] = a[t] + b[t];

        if (t % 100)
            printf("\n%f", c[t]);
    }

}

void kernels_launch(int devices, int multi_gpu) {

    int parts = TOTAL_SIZE / devices;
    int rem = TOTAL_SIZE % devices;
    int i = 0;

    if (multi_gpu) {
        for (i=0; i<devices-1; ++i) {
    
            hipSetDevice(i);
            vec_add<<<parts/block_dim + 1, block_dim, 0, streams[i]>>>(d_c[i], d_a[i], d_b[i], parts);    
        }
    }

    hipSetDevice(i);
    vec_add<<<(parts + rem)/block_dim + 1, block_dim, 0, streams[i]>>>(d_c[i], d_a[i], d_b[i], parts + rem);    

}


void data_transferHtoD(int devices, int multi_gpu) {

    
    int parts = TOTAL_SIZE / devices;
    int rem = TOTAL_SIZE % devices;
    int i = 0;

    if (multi_gpu) {
        for (i=0; i<devices-1; ++i) {
        
            hipSetDevice(i);
            printf("\nS%d", streams[i]);
            chk(hipMemcpyAsync(d_a[i], h_a + (parts * i), sizeof(double) * parts, hipMemcpyHostToDevice, streams[i]));
            chk(hipMemcpyAsync(d_b[i], h_b + (parts * i), sizeof(double) * parts, hipMemcpyHostToDevice, streams[i]));
        }
    }

    hipSetDevice(i);
    chk(hipMemcpyAsync(d_a[i], h_a + (parts * i), sizeof(double) * (parts + rem), hipMemcpyHostToDevice, streams[i]));
    chk(hipMemcpyAsync(d_b[i], h_b + (parts * i), sizeof(double) * (parts + rem), hipMemcpyHostToDevice, streams[i]));

}


void data_transferDtoH(int devices, int multi_gpu) {

    int parts = TOTAL_SIZE / devices;
    int rem = TOTAL_SIZE % devices;
    int i = 0;

    if (multi_gpu) {
        //Data trnsfer back
        for (i=0; i<devices-1; ++i) {
        
            hipSetDevice(i);
            chk(hipMemcpyAsync(h_c + (parts * i), d_c[i], sizeof(double) * parts, hipMemcpyDeviceToHost, streams[i]));
        
        }
        hipSetDevice(i);
        chk(hipMemcpyAsync(h_c + (parts * i), d_c[i], sizeof(double) * (parts + rem), hipMemcpyDeviceToHost, streams[i]));
    }
}

void deallocate(int devices) {

    for (int i=0; i<devices; ++i) {

        hipSetDevice(i);
        hipFreeAsync(d_c[i], streams[i]);
        hipFreeAsync(d_a[i], streams[i]);
        hipFreeAsync(d_b[i], streams[i]);
    }

    for (int i=0; i<devices; ++i) {
        hipStreamDestroy(streams[i]);
    }

    free(d_c);
    free(d_a);
    free(d_b);

    //free(h_a);
    //free(h_b);
    //free(h_c);

    //cudaFreeHost(d_c);
    //cudaFreeHost(d_a);
    //cudaFreeHost(d_b);

    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    
    hipEventDestroy(start);
    hipEventDestroy(finish); 

}

void verify() {

    double diff_sq = 0.0;
    double sum_sq  = 0.0;

    for (int i=0; i<TOTAL_SIZE; ++i) {

        sum_sq += h_c[i] * h_c[i];
        diff_sq += (h_c[i] - (h_a[i] + h_b[i])) * (h_c[i] - (h_a[i] + h_b[i]));
    }

    printf("\n%f\t%f\n", h_c[0], h_c[5]);
    printf("\n\nError Rate: %e\n", diff_sq / sum_sq);
}


int main(int argc, char **argv) {
 
    int i, parts, rem, devices = 1;
    float exec_time;
    int multi_gpu = 0;
    if (argc > 1 && strcmp(argv[1], "-m") == 0) {
       multi_gpu = 1; 
    }

    chk(hipGetDeviceCount(&devices));
    printf("\nNum devices available = %d\n", devices);

    if (devices == 0) {
        printf("\nError: No devices found\n");
        exit(1);
    }
    
    if (devices ==1)
        multi_gpu = 0;

    allocate(devices, multi_gpu); 


    //Initialize data
    for (i=0; i<TOTAL_SIZE; ++i) {
        h_a[i] = i + 1;
        h_b[i] = i + 2;
    }


    data_transferHtoD(devices, multi_gpu);
    
    hipEventRecord(start);
    kernels_launch(devices, multi_gpu);
    hipEventRecord(finish);
    
    for (i=0; i<devices; ++i)
        hipStreamWaitEvent(streams[i], finish);


    data_transferDtoH(devices, multi_gpu);

    for (i=0; i<devices; ++i)
        hipStreamSynchronize(streams[i]);

    if (TOTAL_SIZE <= 2048) { 
        verify();
    }

    hipEventElapsedTime(&exec_time, start, finish);
    
    printf("MultiGPU Time = %f", exec_time / 1000);
    deallocate(devices);
    printf("\nFinished.\n");
    return 0;
}
